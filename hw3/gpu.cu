#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "common.h"

#define NUM_THREADS 256

extern double size;

// Spatial binning parameters
#define MAX_PARTICLES_PER_BIN 64
__device__ int d_bins_size;
__device__ int d_bins_count;

//
//  benchmarking program
//

__device__ void apply_force_gpu(particle_t &particle, particle_t &neighbor)
{
    double dx = neighbor.x - particle.x;
    double dy = neighbor.y - particle.y;
    double r2 = dx * dx + dy * dy;
    if( r2 > cutoff*cutoff )
        return;
    //r2 = fmax( r2, min_r*min_r );
    r2 = (r2 > min_r*min_r) ? r2 : min_r*min_r;
    double r = sqrt( r2 );

    //
    //  very simple short-range repulsive force
    //
    double coef = ( 1 - cutoff / r ) / r2 / mass;
    particle.ax += coef * dx;
    particle.ay += coef * dy;
}

// Kernel to assign particles to spatial bins
__global__ void bin_particles_gpu(particle_t * particles, int n, int * bins, int * bin_counts, double size)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= n) return;
    
    particle_t * p = &particles[tid];
    
    // Calculate bin size based on cutoff radius
    int bins_per_side = (int)ceil(size / cutoff);
    int bin_x = (int)(p->x / cutoff);
    int bin_y = (int)(p->y / cutoff);
    bin_x = min(max(bin_x, 0), bins_per_side - 1);
    bin_y = min(max(bin_y, 0), bins_per_side - 1);
    
    int bin_id = bin_y * bins_per_side + bin_x;
    int pos = atomicAdd(&bin_counts[bin_id], 1);
    
    if(pos < MAX_PARTICLES_PER_BIN) {
        bins[bin_id * MAX_PARTICLES_PER_BIN + pos] = tid;
    }
}

__global__ void compute_forces_gpu(particle_t * particles, int n, int * bins, int * bin_counts, double size)
{
    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= n) return;
    
    particles[tid].ax = particles[tid].ay = 0;
    
    particle_t * p = &particles[tid];
    
    // Calculate which bin this particle is in
    int bins_per_side = (int)ceil(size / cutoff);
    int bin_x = (int)(p->x / cutoff);
    int bin_y = (int)(p->y / cutoff);
    
    bin_x = min(max(bin_x, 0), bins_per_side - 1);
    bin_y = min(max(bin_y, 0), bins_per_side - 1);
    
    for(int dy = -1; dy <= 1; dy++) {
        for(int dx = -1; dx <= 1; dx++) {
            int check_x = bin_x + dx;
            int check_y = bin_y + dy;
            if(check_x < 0 || check_x >= bins_per_side || 
               check_y < 0 || check_y >= bins_per_side) continue;
            
            int check_bin = check_y * bins_per_side + check_x;
            int count = bin_counts[check_bin];
            
            // Iterate through particles in this bin but don't apply 
            // the force to itsself 
            for(int i = 0; i < min(count, MAX_PARTICLES_PER_BIN); i++) {
                int neighbor_id = bins[check_bin * MAX_PARTICLES_PER_BIN + i];
                if(neighbor_id != tid) { 
                    apply_force_gpu(particles[tid], particles[neighbor_id]);
                }
            }
        }
    }
}

__global__ void move_gpu (particle_t * particles, int n, double size)
{

    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= n) return;
    
    particle_t * p = &particles[tid];
    //
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    //
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x  += p->vx * dt;
    p->y  += p->vy * dt;
    
    //
    //  bounce from walls
    //
    while( p->x < 0 || p->x > size )
    {
        p->x  = p->x < 0 ? -(p->x) : 2*size-p->x;
        p->vx = -(p->vx);
    }
    while( p->y < 0 || p->y > size )
    {
        p->y  = p->y < 0 ? -(p->y) : 2*size-p->y;
        p->vy = -(p->vy);
    }

}



int main( int argc, char **argv )
{
    // This takes a few seconds to initialize the runtime
    hipDeviceSynchronize();
    
    if( find_option( argc, argv, "-h" ) >= 0 )
    {
        printf( "Options:\n" );
        printf( "-h to see this help\n" );
        printf( "-n <int> to set the number of particles\n" );
        printf( "-o <filename> to specify the output file name\n" );
        return 0;
    }
    
    int n = read_int( argc, argv, "-n", 1000 );

    char *savename = read_string( argc, argv, "-o", NULL );

    FILE *fsave = savename ? fopen( savename, "w" ) : NULL;
    particle_t *particles = (particle_t*) malloc( n * sizeof(particle_t) );
    
    // GPU particle data structure
    particle_t * d_particles;
    hipMalloc((void **) &d_particles, n * sizeof(particle_t));
    
    set_size( n );

    init_particles( n, particles );
    
    // Calculate bin grid size
    int bins_per_side = (int)ceil(size / cutoff);
    int total_bins = bins_per_side * bins_per_side;
    
    // Allocate memory for spatial binning
    int * d_bins;
    int * d_bin_counts;
    hipMalloc((void **) &d_bins, total_bins * MAX_PARTICLES_PER_BIN * sizeof(int));
    hipMalloc((void **) &d_bin_counts, total_bins * sizeof(int));
    
    hipDeviceSynchronize();
    double copy_time = read_timer( );
    
    // Copy the particles to the GPU
    hipMemcpy(d_particles, particles, n * sizeof(particle_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    copy_time = read_timer( ) - copy_time;
    
    //
    //  simulate a number of time steps
    //
    hipDeviceSynchronize();
    double simulation_time = read_timer( );

    for( int step = 0; step < NSTEPS; step++ )
    {
  
        hipMemset(d_bin_counts, 0, total_bins * sizeof(int));
        int blks = (n + NUM_THREADS - 1) / NUM_THREADS;
        bin_particles_gpu <<< blks, NUM_THREADS >>> (d_particles, n, d_bins, d_bin_counts, size);
        compute_forces_gpu <<< blks, NUM_THREADS >>> (d_particles, n, d_bins, d_bin_counts, size);
        
        //
        //  move particles
        //
        move_gpu <<< blks, NUM_THREADS >>> (d_particles, n, size);
        
        //
        //  save if necessary
        //
        if( fsave && (step%SAVEFREQ) == 0 ) {
            // Copy the particles back to the CPU
            hipMemcpy(particles, d_particles, n * sizeof(particle_t), hipMemcpyDeviceToHost);
            save( fsave, n, particles);
        }
    }
    hipDeviceSynchronize();
    simulation_time = read_timer( ) - simulation_time;
    
    printf( "CPU-GPU copy time = %g seconds\n", copy_time);
    printf( "n = %d, simulation time = %g seconds\n", n, simulation_time );
    
    free( particles );
    hipFree(d_particles);
    hipFree(d_bins);
    hipFree(d_bin_counts);
    if( fsave )
        fclose( fsave );
    
    return 0;
}
