#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include "common.h"

#define MAX_PARTICLES_PER_BIN 64

int bin_index(int x, int y, int bins_per_row) {
    return x + y * bins_per_row;
}

int main(int argc, char **argv)
{
    if (find_option(argc, argv, "-h") >= 0)
    {
        printf("Options:\n");
        printf("-h to see this help\n");
        printf("-n <int> to set the number of particles\n");
        printf("-o <filename> to specify the output file name\n");
        return 0;
    }

    int n = read_int(argc, argv, "-n", 1000);
    char *savename = read_string(argc, argv, "-o", NULL);

    FILE *fsave = savename ? fopen(savename, "w") : NULL;
    particle_t *particles = (particle_t *)malloc(n * sizeof(particle_t));
    set_size(n);
    init_particles(n, particles);

    int bins_per_row = (int)ceil(size / cutoff);
    int bin_count = bins_per_row * bins_per_row;

    particle_t **bins = (particle_t **)malloc(bin_count * sizeof(particle_t *));
    int *bin_sizes = (int *)calloc(bin_count, sizeof(int));
    for (int i = 0; i < bin_count; i++)
        bins[i] = (particle_t *)malloc(MAX_PARTICLES_PER_BIN * sizeof(particle_t));

    double simulation_time = read_timer();

    for (int step = 0; step < NSTEPS; step++)
    {
        // Clear bins
        for (int i = 0; i < bin_count; i++)
            bin_sizes[i] = 0;

        // Re-bin particles
        for (int i = 0; i < n; i++)
        {
            int x = (int)(particles[i].x / cutoff);
            int y = (int)(particles[i].y / cutoff);
            int idx = bin_index(x, y, bins_per_row);
            bins[idx][bin_sizes[idx]++] = particles[i];
        }

        // Compute forces
        for (int i = 0; i < n; i++)
        {
            particle_t *p = &particles[i];
            p->ax = p->ay = 0;

            int x = (int)(p->x / cutoff);
            int y = (int)(p->y / cutoff);

            for (int dx = -1; dx <= 1; dx++)
            {
                for (int dy = -1; dy <= 1; dy++)
                {
                    int nx = x + dx;
                    int ny = y + dy;
                    if (nx >= 0 && ny >= 0 && nx < bins_per_row && ny < bins_per_row)
                    {
                        int nidx = bin_index(nx, ny, bins_per_row);
                        for (int j = 0; j < bin_sizes[nidx]; j++)
                            apply_force(*p, bins[nidx][j]);
                    }
                }
            }
        }

        // Move particles
        for (int i = 0; i < n; i++)
            move(particles[i]);

        // Save
        if (fsave && (step % SAVEFREQ) == 0)
            save(fsave, n, particles);
    }

    simulation_time = read_timer() - simulation_time;
    printf("n = %d, simulation time = %g seconds\n", n, simulation_time);

    free(particles);
    for (int i = 0; i < bin_count; i++)
        free(bins[i]);
    free(bins);
    free(bin_sizes);
    if (fsave)
        fclose(fsave);

    return 0;
}
